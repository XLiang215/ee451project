#include "hip/hip_runtime.h"
#include "maths_convolution.h"
#include <hipblas.h>
#include<hip/hip_runtime_api.h>
#include<hip/hip_runtime.h>

__global__ void convolution(float *I, const float* __restrict__ M, float *P,int channels, int width, int height, int TILE_WIDTH, int maskLength)
{
  __shared__ float N_ds[7][7];
  int k;
  int w_l = TILE_WIDTH + maskLength - 1;
  for (k = 0; k < channels; k++) {					// First batch loading
    int dest = threadIdx.y * TILE_WIDTH + threadIdx.x,
       destY = dest / w_l, destX = dest % w_l,
       srcY = blockIdx.y * TILE_WIDTH + destY - maskLength/2,
       srcX = blockIdx.x * TILE_WIDTH + destX - maskLength/2,
       src = (srcY * width + srcX) * channels + k;
    if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
       N_ds[destY][destX] = I[src];
    else
       N_ds[destY][destX] = 0.0;

       for (int iter=1; iter <= (w_l * w_l) / (TILE_WIDTH*TILE_WIDTH); iter++)
      {					// Second batch loading
        dest = threadIdx.y * TILE_WIDTH + threadIdx.x + iter*(TILE_WIDTH * TILE_WIDTH);
          destY = dest / w_l, destX = dest % w_l;
          srcY  = blockIdx.y * TILE_WIDTH + destY - maskLength/2;
          srcX = blockIdx.x * TILE_WIDTH + destX - maskLength/2;
          src = (srcY * width + srcX) * channels + k;
          if (destY < w_l && destX < w_l)
          {
              if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
                  N_ds[destY][destX] = I[src];
              else
                  N_ds[destY][destX] = 0.0;
          }
      }
    __syncthreads();

    float accum = 0;
    int y, x;
    for (y = 0; y < maskLength; y++)
       for (x = 0; x < maskLength; x++)
          accum += N_ds[threadIdx.y + y][threadIdx.x + x] * M[y * maskLength + x];
    y = blockIdx.y * TILE_WIDTH + threadIdx.y;
    x = blockIdx.x * TILE_WIDTH + threadIdx.x;
    if ((y >= (maskLength - 1) / 2) && (x >= (maskLength - 1) / 2) && (y < height - (maskLength - 1) / 2) && (x < width - (maskLength - 1) / 2))
      P[((y-(maskLength - 1) / 2) * maskLength + x-(maskLength - 1) / 2) * channels + k] = accum;
    __syncthreads();
  }
}


Array3Dd convolution(Array3Dd X, const Array2Dd &Ker, string shape) // 采用数组来求卷积，而不是用vector，速度要快10%！
{

	if (shape != "valid" && shape != "full")
	{
		cout << "wrong convolution shape control!" << endl << "convolution() failed!" << endl;
		Array3Dd temp;
		return temp;
	}

	if (X.size() <= 0)
	{
		cout << "Array3Dd is wrong!" << endl << "convolution() failed!" << endl;
		Array3Dd temp;
		return temp;
	}

	int Ker_row = Ker.at(0).size();
	int Ker_col = Ker.size();

	if (shape == "full")
	{
		X.expand_to_full_size(Ker_col, Ker_row);
	}

	int X_page = X.size();
	int X_row = X.at(0).at(0).size();
	int X_col = X.at(0).size();

	int i, j, k;

	if (shape == "valid" && (X_row < Ker_row || X_col < Ker_col))
	{
		cout << "X size is smaller than Ker size!" << endl << "convolution() failed!" << endl;
		Array3Dd temp;
		return temp;
	}

    
	int conv_row = X.at(0).at(0).size() - Ker.at(0).size() + 1; // 创建卷积结果输出变量conv并初始化为0
	int conv_col = X.at(0).size() - Ker.size() + 1;
	Array3Dd convn(X_page, conv_col, conv_row, 0);

	double *arr_X = new double[X_page * X_row * X_col]();
	double *arr_Ker = new double[Ker_row * Ker_col]();

	
	for (i = 0; i < X_page; i++) //vector 转 数组
	{
		for (j = 0; j < X_row; j++)
		{
			for (k = 0; k < X_col; k++)
			{
				
				arr_X[i * (X_row * X_col) + j * X_col + k] = X.at(i).at(k).at(j); // 对arr_X赋值

				
				if ((i == 0) && (j < Ker_row) && (k < Ker_col)) // 对arr_Ker赋值
				{
					arr_Ker[j * Ker_col + k] = Ker.at(Ker_col - 1 - k).at(Ker_row - 1 - j);// x,y向同时翻转
				}
			}
		}
	}


	int maskLength = 5;

    int imageChannels = X_page;
    int imageWidth = X_row;
    int imageHeight = X_col;

    int TILE_WIDTH = 3;

	float * hostOutputImageData;

	float * deviceInputImageData;
	float * deviceOutputImageData;
	float * deviceMaskData;

	hostOutputImageData = (float *) malloc(sizeof(float)*(imageWidth - maskLength + 1)*(imageHeight - maskLength + 1)*imageChannels);
		
	hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
	hipMalloc((void **) &deviceOutputImageData, (imageWidth - maskLength + 1) * (imageHeight - maskLength + 1) * imageChannels * sizeof(float));
	hipMalloc((void **) &deviceMaskData, maskLength * maskLength * sizeof(float));

	hipMemcpy(deviceInputImageData, //copy image to device
		     arr_X,
		     imageWidth * imageHeight * imageChannels * sizeof(float),
		     hipMemcpyHostToDevice);
	hipMemcpy(deviceMaskData,       //copy mask to device
		     arr_Ker,
		     maskLength * maskLength * sizeof(float),
		     hipMemcpyHostToDevice);

    
    dim3 dimGrid(((imageWidth-1)/TILE_WIDTH)+1, ((imageHeight-1)/TILE_WIDTH)+1,1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	convolution<<<dimGrid,dimBlock>>>(deviceInputImageData, deviceMaskData, deviceOutputImageData, 
										imageChannels, imageWidth, imageHeight, TILE_WIDTH, maskLength);
    
	
    hipMemcpy(hostOutputImageData, //copy result to host
	         deviceOutputImageData,
	         (imageWidth - maskLength + 1) * (imageHeight - maskLength + 1) * imageChannels * imageChannels * sizeof(float),
	         hipMemcpyDeviceToHost);

    int id = 0;
    for (i = 0; i < X_page; i++) {
		for (j = 0; j < conv_row; j++) {
			for (k = 0; k < conv_col; k++) {
				convn.at(i).at(k).at(j) = hostOutputImageData[id++];
			}
		}
	}

	delete[] arr_X;
	delete[] arr_Ker;
	free(hostOutputImageData);
	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);
	hipFree(deviceMaskData);

	return convn;
}



Array2Dd convolution(const Array3Dd &X, const Array3Dd &Ker, string shape) 
{

	if (shape != "valid" && shape != "full")
	{
		cout << "wrong convolution shape control!" << endl << "convolution() failed!" << endl;
		Array2Dd temp;
		return temp;
	}

	int page_X = X.size();
	int page_Ker = Ker.size();

	if (page_X != page_Ker)
	{
		cout << "page size not equal!" << endl << "convolution() failed!" << endl;
		Array2Dd temp;
		return temp;
	}

	Array2Dd sum;

	for (int i = 0; i < page_X; ++i)
	{
		sum.add(convolution(X.at(i), Ker.at(i), shape));
	}

	return sum;
}


Array2Dd convolution(Array2Dd X, Array2Dd Ker, string shape) // 采用数组来求卷积，而不是用vector，速度要快30倍！
{
	if (shape != "valid" && shape != "full")
	{
		cout << "wrong convolution shape control!" << endl << "convolution() failed!" << endl;
		Array2Dd temp;
		return temp;
	}

	int Ker_row = Ker.at(0).size();
	int Ker_col = Ker.size();

	if (shape == "full")
	{
		X.expand_to_full_size(Ker_col, Ker_row);
	}

	int X_row = X.at(0).size();
	int X_col = X.size();

	if (shape == "valid" && (X_row < Ker_row || X_col < Ker_col))
	{
		cout << "X size is smaller than Ker size!" << endl << "convolution() failed!" << endl;
		Array2Dd temp;
		return temp;
	}

   
	int conv_row = X.at(0).size() - Ker.at(0).size() + 1;  // 创建卷积结果输出变量conv并初始化为0
	int conv_col = X.size() - Ker.size() + 1;
	Array2Dd conv(conv_col, conv_row, 0);

	double *arr_X = new double[X_row * X_col]();
	double *arr_Ker = new double[Ker_row * Ker_col]();

	int i, j;

	for (i = 0; i < X_row; i++)
	{
		for (j = 0; j < X_col; j++)
		{	
			arr_X[i * X_col + j] = X.at(j).at(i); // 对arr_X赋值  
			if ((i < Ker_row) && (j < Ker_col)) // 对arr_Ker赋值
			{
				arr_Ker[i * Ker_col + j] = Ker.at(Ker_col - 1 - j).at(Ker_row - 1 - i); // x,y向同时翻转
			}
		}
	}

	int row, col;
	for (i = 0; i < conv_row; i++)
	{
		for (j = 0; j < conv_col; j++)
		{
            double sum_ij = 0; // 计算卷积矩阵第(i,j)点的值
			for (row = i; row < i + Ker_row; row++)
			{
				for (col = j; col < j + Ker_col; col++)
				{
					sum_ij += arr_X[row * X_col + col] * arr_Ker[(row - i) * Ker_col + (col - j)];
				}
			}
			conv.at(j).at(i) = sum_ij;
		}
	}

	delete[] arr_X;
	delete[] arr_Ker;

	return conv;
}

